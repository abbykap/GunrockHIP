#include "hip/hip_runtime.h"
#include <gunrock/algorithms/ppr.hxx>
#include "ppr_cpu.hxx"

using namespace gunrock;
using namespace memory;

void test_ppr(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;
  csr_t csr;

  // --
  // IO

  weight_t alpha = 0.15;
  weight_t epsilon = 1e-6;
  vertex_t n_seeds = 10;

  std::string filename = argument_array[1];
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto [properties, coo] = mm.load(filename);

  if (util::is_market(filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    csr.from_coo(coo);
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  // --
  // Build graph

  auto G = graph::build<memory_space_t::device>(properties, csr);

  // --
  // Params and memory allocation

  vertex_t n_vertices = G.get_number_of_vertices();

  thrust::device_vector<weight_t> p(n_seeds * n_vertices);

  // --
  // GPU Run

  float gpu_elapsed =
      gunrock::ppr::run_batch(G, n_seeds, p.data().get(), alpha, epsilon);

  // --
  // CPU Run

  thrust::host_vector<weight_t> h_p(n_seeds * n_vertices);

  float cpu_elapsed = ppr_cpu::run<csr_t, vertex_t, edge_t, weight_t>(
      csr, n_seeds, h_p.data(), alpha, epsilon);

  int n_errors = util::compare(p.data().get(), h_p.data(), n_seeds * n_vertices,
                               [epsilon](const weight_t a, const weight_t b) {
                                 return std::abs(a - b) > epsilon;
                               });

  // --
  // Log + Validate

  print::head(p, 40, "GPU rank");
  print::head(h_p, 40, "CPU rank");

  std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
  std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  std::cout << "Number of errors : " << n_errors << std::endl;
}

int main(int argc, char** argv) {
  test_ppr(argc, argv);
}
